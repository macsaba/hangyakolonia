
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <float.h>
#include <string.h>
// Kernel function to add the elements of two arrays

#define ALPHA 5.1
#define BETA 0.15
#define RHO 0.6
#define Q 5.0

__global__
void calcDenom(int N, float *p, float *tau, float *dist,float *denominator, int *connections)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for(int r = index; r < N; r += stride)
  {
    //probability denominator
    denominator[r] = 0;
    for(int s = 0; s < N; s++)
    {
        denominator[r] += pow(tau[r*N + s], ALPHA)*pow((1/dist[r*N + s]), BETA)*connections[r*N + s];
    }
  }
}
__global__
void calcProb(int N, float *p, float *tau, float *dist,float *denominator, int *connections)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for(int r = index; r < N*N; r += stride)
  {
    if(denominator[(int) r/N] != 0)
    {
      p[r] = (pow(tau[r], ALPHA)*pow((1/dist[r]), BETA)*connections[r])/denominator[(int) r/N];
    }
    else
    {
      p[r] = -1;
    }

  }
}
__global__
void setZeros(int N, float *p, float *tau,float *dtau, float *dist,float *denominator, int *connections)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for(int r = index; r < N*N; r += stride)
  {
    connections[r] = 0;
    tau[r] = 1.0;
    dtau[r] = 0;
    dist[r] = 1.0;
    p[r] = 0;
  }
}

__global__
void clearTau(int N, float *tau,float *dtau)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for(int r = index; r < N*N; r += stride)
  {
    tau[r] = dtau[r] + (1-RHO)*tau[r] +1;
    dtau[r] = 0;
  }
}

void read_csv(int row, int col, char *filename, double **data){
	FILE *file;
	file = fopen(filename, "r");

	int i = 0;
  char line[100];

	while (fgets(line, 4098, file) && (i < row))
	  {
	    char* tmp = strdup(line);
	    int j = 0;

	    char *token;

	    /* get the first token */
	    token = strtok(line, " ");

	    /* walk through other tokens */
	    while( token != NULL ) {
	      data[i][j] = atof(token);
	      token = strtok(NULL, " ");
	      j++;
	    }
	    free(tmp);
	    i++;
	  }

}

void readEdges(int N, int row, int col, char *filename, int *connections, float *dist){
  double **dataEdges;
  dataEdges = (double **)malloc(row * sizeof(double *));
  for (int i = 0; i < row; ++i){
    dataEdges[i] = (double *)malloc(col * sizeof(double));
  }

  read_csv(row, col, filename, dataEdges);

  for(int i = 0; i < row; i++)
  {
      connections[((int) dataEdges[i][1])*N + (int) dataEdges[i][2]] = 1;
      dist[(int) dataEdges[i][1] *N + (int) dataEdges[i][2]] = dataEdges[i][3];
  }

  free(dataEdges);

}

int main(void)
{
  //CONSTANTS
  const int NR_OF_DATAPOINTS = 1363;
  const int NR_OF_EDGES = 3977;
  const int NR_OF_ITERATIONS = 5;
  const int NR_OF_ANTS = 20;

  const int NR_OF_BLOCKS = 4096;
  const int NR_OF_THREADS = 512;
  const int start = 1;
  const int destination = 250;

  //MEMORY ALLOCATION
  float *tau, *dtau, *dist, *p;
  int *connections, *numOfConnections;
  float *denominator;
  hipMallocManaged(&tau, NR_OF_DATAPOINTS*NR_OF_DATAPOINTS*sizeof(float));
  hipMallocManaged(&dtau, NR_OF_DATAPOINTS*NR_OF_DATAPOINTS*sizeof(float));
  hipMallocManaged(&dist, NR_OF_DATAPOINTS*NR_OF_DATAPOINTS*sizeof(float));
  hipMallocManaged(&p, NR_OF_DATAPOINTS*NR_OF_DATAPOINTS*sizeof(float));
  hipMallocManaged(&connections, NR_OF_DATAPOINTS*NR_OF_DATAPOINTS*sizeof(int));
  hipMallocManaged(&denominator, NR_OF_DATAPOINTS*sizeof(float));
  hipMallocManaged(&numOfConnections, NR_OF_DATAPOINTS*sizeof(int));

  setZeros<<<NR_OF_BLOCKS, NR_OF_THREADS>>>(NR_OF_DATAPOINTS, p, tau, dtau, dist,denominator, connections);
  hipDeviceSynchronize();

  int route [NR_OF_ANTS][200];
  float length [NR_OF_ANTS];
  float lengthMin [NR_OF_ANTS];
  //READ EDGES
  int col     = 6;
  char fname[256] = "dist.csv";

  readEdges(NR_OF_DATAPOINTS, NR_OF_EDGES, col, fname, connections, dist);

  //count the connections of nodes
  for(int r = 0; r < NR_OF_DATAPOINTS; r++)
  {
      numOfConnections[r] = 0;
      for(int s = 0; s < NR_OF_DATAPOINTS; s++)
      {
          numOfConnections[r] += connections[NR_OF_DATAPOINTS*r + s];
      }
  }

  //iteration
  float roulette = 0;
  float sumP = 0;
  int selection = 0;
  int current = 0;
  int step = 0;


  for(int i = 0; i < NR_OF_ITERATIONS; i++)
  {
    clearTau<<<NR_OF_BLOCKS, NR_OF_THREADS>>>(NR_OF_DATAPOINTS, tau, dtau);
    hipDeviceSynchronize();
    //"create ants"
    for(int ant = 0; ant < NR_OF_ANTS; ant++)
    {
        route[ant][0] = start;
        length[ant] = 0;
        lengthMin[ant] = FLT_MAX;

        calcDenom<<<NR_OF_BLOCKS, NR_OF_THREADS>>>(NR_OF_DATAPOINTS, p, tau, dist,denominator, connections);
        hipDeviceSynchronize();

        //calculate probabilities
        calcProb<<<NR_OF_BLOCKS, NR_OF_THREADS>>>(NR_OF_DATAPOINTS, p, tau, dist,denominator, connections);
        hipDeviceSynchronize();

        //start an ant
        step = 0;
        while(step < 200)
        {
            //ROULETTE
            current = route[ant][step];

            do
            {
              roulette = (double) rand() / (double) RAND_MAX;
              sumP = 0;
              selection = -1;
              for(int s = 0; s < NR_OF_DATAPOINTS; s++)
              {

                  sumP += p[route[ant][step]*NR_OF_DATAPOINTS + s];
                if(roulette < sumP)
                  {
                      selection = s;
                      break;
                  }
              }
            }while (selection == route[ant][step - 1] && numOfConnections[current] != 1);

            if(selection == destination)
            {
                printf("\ntalalat\n");

                for(int ii = 0; ii <=step ; ii++)
                {
                  printf("-> %d ", route[ant][ii]);
                }
                printf("\n\n");
                step++;
                route[ant][step] = selection;
                length[ant] += dist[route[ant][step - 1]*NR_OF_DATAPOINTS + route[ant][step]];
                break;
            }

            if(numOfConnections[selection] <= 1)
            {
                connections[current*NR_OF_DATAPOINTS + selection] = 0;
                connections[selection*NR_OF_DATAPOINTS + current] = 0;
                numOfConnections[current] = numOfConnections[current] - numOfConnections[selection];
                numOfConnections[selection] = 0;
            }
            else
            {
                //step forward
                step++;
                route[ant][step] = selection;
                length[ant] += dist[route[ant][step - 1]*NR_OF_DATAPOINTS + route[ant][step]];
                //feromon
                dtau[current*NR_OF_DATAPOINTS + selection] += Q/length[ant];

            }
        }
    }
  }

  hipFree(denominator);
  hipFree(tau);
  hipFree(dtau);
  hipFree(dist);
  hipFree(p);
  hipFree(connections);
  hipFree(numOfConnections);

}
